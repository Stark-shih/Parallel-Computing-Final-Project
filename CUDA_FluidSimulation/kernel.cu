#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <SFML/Graphics.hpp>
#include <iostream>
#include <cmath>
#include <assert.h>
#include <iomanip>
#include <stdio.h>

using namespace sf;

//solver.h
class Solver
{
private:
    /* cuda */
    int numberofblocks;
    int numberofthreads;
    /* data */
    int screenWidth;
    int screenHeight;
    int gridSizeX;
    int gridSizeY;
    int deviceId;

    float minX;
    float minY;
    float maxX;
    float maxY;
    float viscosity;

    float4* u;
    float4* tmp;
    float4* div;
    float4* p;
public:
    Solver(int width, int height, int resolution);
    ~Solver();
    void reset();
    void update(float dt, float2 forceOrigin, float2 forceVector, Uint8* pixels);
    void print(float4* matrix);
};
//solver.cpp
Solver::Solver(int screenWidth, int screenHeight, int resolution)
{
    assert((resolution * screenHeight) % screenWidth == 0);
    this->screenWidth = screenWidth;
    this->screenHeight = screenHeight;
    gridSizeX = resolution;
    gridSizeY = resolution * screenHeight / screenWidth;
    minX = 1.0f;
    minY = 1.0f;
    maxX = gridSizeX - 1.0f;
    maxY = gridSizeY - 1.0f;
    viscosity = 0.005;
}

Solver::~Solver()
{
}

void Solver::reset() {
    int numberOfSMs;
    hipGetDevice(&(this->deviceId));
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    this->numberofblocks = 16 * numberOfSMs;
    this->numberofthreads = 128;

    hipMallocManaged(&(this->u), gridSizeY * gridSizeX * sizeof(float4));
    hipMallocManaged(&(this->tmp), gridSizeY * gridSizeX * sizeof(float4));
    hipMallocManaged(&(this->div), gridSizeY * gridSizeX * sizeof(float4));
    hipMallocManaged(&(this->p), gridSizeY * gridSizeX * sizeof(float4));
}

void Solver::print(float4* matrix) {
    for (int i = 0; i < gridSizeY; i++) {
        for (int j = 0; j < gridSizeX; j++) {
            float amp = sqrtf(matrix[i * gridSizeX + j].x * matrix[i * gridSizeX + j].x + matrix[i * gridSizeX + j].y * matrix[i * gridSizeX + j].y);
            std::cout << std::fixed << std::setprecision(0) << amp;
        }
        std::cout << "\n";
    }
}


void swap(float4*& field1, float4*& field2) {
    float4* temp = field1;
    field1 = field2;
    field2 = temp;
}
// __device__
float clampTo_0_255(float val) {
	if (val < 0.f) val = 0;
	if (val > 255.0f) val = 255;
	return val;
}
__global__
void setBoundary(float4* field, float sc, int w, int h) {
    /* horizontal: the first line and the last line */
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int j = index; j < w; j += stride) {
        field[j] = make_float4(sc * field[w + j].x, sc * field[w + j].y, sc * field[w + j].z, sc * field[w + j].w);
        field[(h - 1) * w + j] = make_float4(sc * field[(h - 2) * w + j].x, sc * field[(h - 2) * w + j].y, sc * field[(h - 2) * w + j].z, sc * field[(h - 2) * w + j].w);
    }
    /* vetrtical */
    for (int i = index; i < h; i += stride) {
        field[i * w] = make_float4(sc * field[i * w + 1].x, sc * field[i * w + 1].y, sc * field[i * w + 1].z, sc * field[i * w + 1].w);
        field[i * w + w - 1] = make_float4(sc * field[i * w + w - 2].x, sc * field[i * w + w - 2].y, sc * field[i * w + w - 2].z, sc * field[i * w + w - 2].w);
    }
}
__global__
void cuda_addForce(int gridSizeX, int gridSizeY, float2 forceOrigin, float2 forceVector, float4* w_in, float4* w_out) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;
        float2 pos = make_float2(b, a);

        float distance = sqrtf((pos.x - forceOrigin.x) * (pos.x - forceOrigin.x) + (pos.y - forceOrigin.y) * (pos.y - forceOrigin.y));
        float amp = exp(-distance/10);
        w_out[a * gridSizeX + b].x = (w_in[a * gridSizeX + b].x + forceVector.x * amp);
        w_out[a * gridSizeX + b].y = (w_in[a * gridSizeX + b].y + forceVector.y * amp);
    }
}
__global__
void cuda_advect(int gridSizeX, int gridSizeY, float dt, float4* u, float4* xNew) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    float rdx, rdy, oldx, oldy, dx, dy, mdx, mdy;
    int xid0, xid1, yid0, yid1;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;
        oldx = (b+0.5) - dt * u[a * gridSizeX + b].x * gridSizeX;
        oldy = (a+0.5) - dt * u[a * gridSizeY + b].y * gridSizeX;
        oldx = fmax(0.5f, fmin(gridSizeX-0.5f, oldx));
        oldy = fmax(0.5f, fmin(gridSizeY-0.5f, oldy));
        rdx = round(oldx);
        rdy = round(oldy);
        xid0 = (int)(rdx - 0.5);
        xid1 = xid0 + 1;
        yid0 = (int)(rdy - 0.5);
        yid1 = yid0 + 1;
        dx = oldx - (rdx - 0.5);
        mdx = (rdx + 0.5) - oldx;
        dy = oldy - (rdy - 0.5);
        mdy = (rdy + 0.5) - oldy;
        xNew[a * gridSizeX + b].x = mdx * (mdy * u[yid0 * gridSizeX + xid0].x + dy * u[yid1 * gridSizeX + xid0].x) + dx * (mdy * u[yid0 * gridSizeX + xid1].x + dy * u[yid1 * gridSizeX + xid1].x);
        xNew[a * gridSizeX + b].y = mdx * (mdy * u[yid0 * gridSizeX + xid0].y + dy * u[yid1 * gridSizeX + xid0].y) + dx * (mdy * u[yid0 * gridSizeX + xid1].y + dy * u[yid1 * gridSizeX + xid1].y);
    
    }
}
__global__
void cuda_divergence(int gridSizeX, int gridSizeY, float4* w, float4* div, float4* p) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;

        float wL = w[a * gridSizeX + b - 1].x;
        float wR = w[a * gridSizeX + b + 1].x;
        float wT = w[(a - 1) * gridSizeX + b].y;
        float wB = w[(a + 1) * gridSizeX + b].y;
        div[a * gridSizeX + b].w = -0.5 * ((wR - wL) + (wB - wT)) / gridSizeX;
        p[a * gridSizeX + b] = make_float4(0,0,0,0);
    }
}
__global__
void cuda_jacobi(int gridSizeX, int gridSizeY,  float alpha, float beta, float4* x, float4* b_, float4* xNew) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;

        float4 xL = x[a * gridSizeX + b - 1];
        float4 xR = x[a * gridSizeX + b + 1];
        float4 xT = x[(a - 1) * gridSizeX + b];
        float4 xB = x[(a + 1) * gridSizeX + b];
        float4 bc = b_[a * gridSizeX + b];
        xNew[a * gridSizeX + b].z = ((xL.z + xR.z + xT.z + xB.z)*alpha + bc.w) / beta;
    }
}
__global__
void cuda_subgradient(int gridSizeX, int gridSizeY, float4* p, float4* w, float4* uNew) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;

        float4 pL = p[a * gridSizeX + b - 1];
        float4 pR = p[a * gridSizeX + b + 1];
        float4 pT = p[(a - 1) * gridSizeX + b];
        float4 pB = p[(a + 1) * gridSizeX + b];

        uNew[a * gridSizeX + b] = w[a * gridSizeX + b];
        uNew[a * gridSizeX + b].x -= 0.5 * (pR.z - pL.z) * gridSizeX;
        uNew[a * gridSizeX + b].y -= 0.5 * (pB.z - pT.z) * gridSizeX;
    }
}
__global__ 
void cuda_print(float4* u) {
    printf("%f\n", u[200*400+200].z);
}
//adect->forceaply->applyDye->divergence->jacobiviscousdiffusion->applygradient
void Solver::update(float dt, float2 forceOrigin, float2 forceVector, Uint8* pixels) {

    // external force
    cuda_addForce<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY, forceOrigin, forceVector, u, tmp);
    swap(tmp, u);
    setBoundary<<< numberofblocks, numberofthreads >>>(u, -1.0f, gridSizeX, gridSizeY);
    // diffussion
    for (int s = 0; s < 10; s++) {
        cuda_jacobi<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY, dt*viscosity*gridSizeX*gridSizeY, 1+4*dt*viscosity*gridSizeX*gridSizeY, u, u, tmp);
        cuda_jacobi<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY, dt*viscosity*gridSizeX*gridSizeY, 1+4*dt*viscosity*gridSizeX*gridSizeY, tmp, tmp, u);
        setBoundary<<< numberofblocks, numberofthreads >>>(u, -1.0f, gridSizeX, gridSizeY);
    }
    // -------------------- projection start----------------------
    // divergence
    cuda_divergence<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY,  u, div, p);
    setBoundary<<<numberofblocks, numberofthreads >>>(div, 1.0f, gridSizeX, gridSizeY);
    // pressure
    for (int s = 0; s < 20; s++) {
        cuda_jacobi<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY, 1, 4, p, div, tmp);
        cuda_jacobi<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY, 1, 4, tmp, div, p);
        setBoundary<<< numberofblocks, numberofthreads >>>(p, 1.0f, gridSizeX, gridSizeY);
    }
    // subGradient
    cuda_subgradient<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY, p, u, tmp);
    swap(tmp, u);
    setBoundary<<< numberofblocks, numberofthreads >>>(u, -1.0f, gridSizeX, gridSizeY);
    // -------------------- projection end ----------------------
    // advect
    cuda_advect<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY, dt, u, tmp);
    swap(tmp, u);
    setBoundary<<< numberofblocks, numberofthreads >>>(u, -1.0f, gridSizeX, gridSizeY);
    // -------------------- projection start----------------------
    // divergence
    cuda_divergence<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY,  u, div, p);
    setBoundary<<<numberofblocks, numberofthreads >>>(div, 1.0f, gridSizeX, gridSizeY);
    // pressure
    for (int s = 0; s < 20; s++) {
        cuda_jacobi<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY, 1, 4, p, div, tmp);
        cuda_jacobi<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY, 1, 4, tmp, div, p);
        setBoundary<<< numberofblocks, numberofthreads >>>(p, 1.0f, gridSizeX, gridSizeY);
    }
    // subGradient
    cuda_subgradient<<< numberofblocks, numberofthreads >>>(gridSizeX, gridSizeY, p, u, tmp);
    swap(tmp, u);
    setBoundary<<< numberofblocks, numberofthreads >>>(u, -1.0f, gridSizeX, gridSizeY);
    // -------------------- projection end ----------------------

    // cuda_print<<< 1,1 >>>(p);
    //finish
    hipDeviceSynchronize();
    hipMemPrefetchAsync(u, gridSizeY * gridSizeX * sizeof(float4), deviceId);

    // apply color
    for (int i = 0; i < gridSizeY; i++) {
        for (int j = 0; j < gridSizeX; j++) {
            pixels[(i * gridSizeX + j) * 4] = 138;
            pixels[(i * gridSizeX + j) * 4 + 1] = 43;
            pixels[(i * gridSizeX + j) * 4 + 2] = 226;
            float amp = sqrtf(u[i*gridSizeX+ j].x * u[i * gridSizeX + j].x + u[i * gridSizeX + j].y * u[i * gridSizeX + j].y) * 200;
            pixels[(i * gridSizeX + j) * 4 + 3] = (int) clampTo_0_255(amp);
        }
    }
}

//main.cpp
int main()
{
    int W = 400, H = 400;
    RenderWindow window(VideoMode(W, H), "stable fluid");
    //window.setFramerateLimit(60);

    Uint8* pixels = new Uint8[W * H * 4];
    Texture texture;
    texture.create(W, H);
    Sprite sprite(texture);
    for (register int i = 0; i < W * H * 4; i += 4) {
        pixels[i] = 0;
        pixels[i + 1] = 0;
        pixels[i + 2] = 0;
        pixels[i + 3] = 255;
    }

    Clock clock;
    Time t;
    Vector2i last_pos, now_pos;
    float2 forceVector = make_float2(0, 0);
    float2 forceOrigin = make_float2(0, 0);
    bool click_flag = false;
    Solver stableSolver(W, H, W);
    stableSolver.reset();
    float  timestep = 0.01;
    while (window.isOpen())
    {
        Event event;
        while (window.pollEvent(event))
        {
            switch (event.type) {
            case Event::Closed:
                window.close();
                break;
            case Event::Event::MouseButtonReleased:
                click_flag = false;
                forceOrigin = make_float2(0, 0);
                forceVector = make_float2(0, 0);
                break;
            case Event::MouseButtonPressed:
                click_flag = true;
                last_pos = Mouse::getPosition(window);         
                break;
            case Event::MouseMoved:
                if (click_flag) {
                    now_pos = Mouse::getPosition(window);
                    forceOrigin = make_float2(last_pos.x, last_pos.y);
                    forceVector = make_float2(now_pos.x - last_pos.x, now_pos.y - last_pos.y);
                    last_pos = now_pos;
                }
                break;
            default:
                break;
            }

        }


        float elapsed = clock.getElapsedTime().asSeconds();
        if (elapsed > timestep) {
            stableSolver.update(timestep, forceOrigin, forceVector, pixels);
            clock.restart();
        }


        texture.update(pixels);
        window.clear();
        window.draw(sprite);
        window.display();
    }

    return 0;
}
