#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <SFML/Graphics.hpp>
#include <iostream>
#include <cmath>
#include <assert.h>
#include <iomanip>
#include <stdio.h>

using namespace sf;

//solver.h
class Solver
{
private:
    /* cuda */
    int numberofblocks = 20;
    int numberofthreads = 20;
    /* data */
    int screenWidth;
    int screenHeight;
    int gridSizeX;
    int gridSizeY;

    float minX;
    float minY;
    float maxX;
    float maxY;
    float viscosity;

    float4* u;
    float4* tmp;
public:
    Solver(int width, int height, int resolution);
    ~Solver();
    void reset();
    void update(float dt, float2 forceOrigin, float2 forceVector, Uint8* pixels);
    void print(float4* matrix);
};
//solver.cpp
Solver::Solver(int screenWidth, int screenHeight, int resolution)
{
    assert((resolution * screenHeight) % screenWidth == 0);
    this->screenWidth = screenWidth;
    this->screenHeight = screenHeight;
    gridSizeX = resolution;
    gridSizeY = resolution * screenHeight / screenWidth;
    minX = 1.0f;
    minY = 1.0f;
    maxX = gridSizeX - 1.0f;
    maxY = gridSizeY - 1.0f;
    viscosity = 1e-6f;
}

Solver::~Solver()
{
}

void Solver::reset() {
    int deviceId;
    int numberOfSMs;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    this->numberofblocks = 16 * numberOfSMs;
    this->numberofthreads = 128;

    hipMallocManaged(&(this->u), gridSizeY * gridSizeX * sizeof(float4));
    hipMallocManaged(&(this->tmp), gridSizeY * gridSizeX * sizeof(float4));

    hipMemset(this->u, 0, 1000 * gridSizeY * gridSizeX * sizeof(float4));
    hipMemset(this->tmp, 0, 1000 * gridSizeY * gridSizeX * sizeof(float4));
}
void swap(float4*& field1, float4*& field2) {
    float4* temp = field1;
    field1 = field2;
    field2 = temp;
}
__global__
void setBoundary(float4* field, float sc, int w, int h) {
    /* horizontal: the first line and the last line */
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int j = index; j < w; j += stride) {
        field[j] = make_float4(sc * field[w + j].x, sc * field[w + j].y, sc * field[w + j].z, sc * field[w + j].w);
        field[(h - 1) * w + j] = make_float4(sc * field[(h - 2) * w + j].x, sc * field[(h - 2) * w + j].y, sc * field[(h - 2) * w + j].z, sc * field[(h - 2) * w + j].w);
    }
    /* vetrtical */
    for (int i = index; i < h; i += stride) {
        field[i * w] = make_float4(sc * field[i * w + 1].x, sc * field[i * w + 1].y, sc * field[i * w + 1].z, sc * field[i * w + 1].w);
        field[i * w + w - 1] = make_float4(sc * field[i * w + w - 2].x, sc * field[i * w + w - 2].y, sc * field[i * w + w - 2].z, sc * field[i * w + w - 2].w);
    }
}
__global__
void cuda_addForce(int gridSizeX, int gridSizeY, float2 forceOrigin, float2 forceVector, float4* w_in, float4* w_out) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        float2 pos = make_float2(a, b);

        float distance = sqrtf((pos.x - forceOrigin.x) * (pos.x - forceOrigin.x) + (pos.y - forceOrigin.y) * (pos.y - forceOrigin.y));
        float amp = exp(-distance);
        w_out[a * gridSizeX + b].x = w_in[a * gridSizeX + b].x + forceVector.x * amp;
        w_out[a * gridSizeX + b].y = w_in[a * gridSizeX + b].y + forceVector.y * amp;
    }
}
__global__
void cuda_advect(int gridSizeX, int gridSizeY, float dt, float4* u, float4* xNew) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    float oldx, oldy, dx, dy, mdx, mdy;
    int xid0, xid1, yid0, yid1;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        oldx = a - dt * u[a * gridSizeX + b].x;
        oldy = b - dt * u[a * gridSizeY + b].y;
        oldx = fmax(0.5f, fmin(gridSizeX + 0.5f, oldx));
        oldy = fmax(0.5f, fmin(gridSizeY + 0.5f, oldy));
        xid0 = (int)oldx;
        xid1 = xid0 + 1;
        yid0 = (int)oldy;
        yid1 = yid0 + 1;
        dx = oldx - xid0;
        mdx = 1 - xid0;
        dy = oldy - yid0;
        mdy = 1 - dy;
        xNew[a * gridSizeX + b].x = mdx * (mdy * u[xid0 * gridSizeX + yid0].x + dy * u[xid0 * gridSizeX + yid1].x) + dx * (mdy * u[xid1 * gridSizeX + yid0].x + dy * u[xid1 * gridSizeX + yid1].x);
        xNew[a * gridSizeX + b].y = mdx * (mdy * u[xid0 * gridSizeX + yid0].y + dy * u[xid0 * gridSizeX + yid1].y) + dx * (mdy * u[xid1 * gridSizeX + yid0].y + dy * u[xid1 * gridSizeX + yid1].y);
        xNew[a * gridSizeX + b].z = u[xid0 * gridSizeX + yid0].z;
        xNew[a * gridSizeX + b].w = 1.0;
    }
}
__global__
void cuda_divergence(int gridSizeX, int gridSizeY, float4* w, float4* div) {//gridSizeX, gridSizeY, 0.5 / dx, u, div
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;
        float2 pos = make_float2(a + 0.5, b + 0.5);
        a = (int)pos.x;
        b = (int)pos.y;

        float wL = w[a * gridSizeX + b - 1].x;
        float wR = w[a * gridSizeX + b + 1].x;
        float wT = w[(a - 1) * gridSizeX + b].y;
        float wB = w[(a + 1) * gridSizeX + b].y;
        div[a * gridSizeX + b].x = w[a * gridSizeX + b].x;
        div[a * gridSizeX + b].y = w[a * gridSizeX + b].y;
        div[a * gridSizeX + b].z = w[a * gridSizeX + b].z;
        div[a * gridSizeX + b].w = 0.5 * ((wR - wL) + (wT - wB));
    }
}
__global__
void cuda_jacobi(int gridSizeX, int gridSizeY, float alpha, float rbeta, float4* x, float4* xNew) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;
        float2 pos = make_float2(a + 0.5, b + 0.5);
        a = (int)pos.x;
        b = (int)pos.y;

        float4 xL = x[a * gridSizeX + b - 1];
        float4 xR = x[a * gridSizeX + b + 1];
        float4 xT = x[(a - 1) * gridSizeX + b];
        float4 xB = x[(a + 1) * gridSizeX + b];
        xNew[a * gridSizeX + b].x = x[a * gridSizeX + b].x;
        xNew[a * gridSizeX + b].y = x[a * gridSizeX + b].y;
        xNew[a * gridSizeX + b].z = (xL.z + xR.z + xT.z + xB.z + x[a * gridSizeX + b].w * alpha) * rbeta;;
        xNew[a * gridSizeX + b].w = x[a * gridSizeX + b].w;
    }
}
__global__
void cuda_subgradient(int gridSizeX, int gridSizeY, float4* u, float4* uNew) {//gridSizeX, gridSizeY, 0.5 / dx, p, u, tmp
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;
        float2 pos = make_float2(a + 0.5, b + 0.5);
        a = (int)pos.x;
        b = (int)pos.y;

        float4 pL = u[a * gridSizeX + b - 1];
        float4 pR = u[a * gridSizeX + b + 1];
        float4 pT = u[(a - 1) * gridSizeX + b];
        float4 pB = u[(a + 1) * gridSizeX + b];

        uNew[a * gridSizeX + b] = u[a * gridSizeX + b];
        uNew[a * gridSizeX + b].x -= 0.5 * (pR.z - pL.z);
        uNew[a * gridSizeX + b].y -= 0.5 * (pB.z - pT.z);
    }
}
//adect->forceaply->applyDye->divergence->jacobiviscousdiffusion->applygradient
void Solver::update(float dt, float2 forceOrigin, float2 forceVector, Uint8* pixels) {

    // external force
    cuda_addForce << <numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, forceOrigin, forceVector, u, tmp);
    swap(tmp, u);
    setBoundary << <numberofblocks, numberofthreads >> > (u, -1.0f, gridSizeX, gridSizeY);
    // advect
    cuda_advect << <numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, dt, u, tmp);
    swap(tmp, u);
    setBoundary << <numberofblocks, numberofthreads >> > (u, -1.0f, gridSizeX, gridSizeY);
    //divergence
    cuda_divergence << <numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, u, tmp); // u -> div
    swap(tmp, u);
    setBoundary << <numberofblocks, numberofthreads >> > (u, -1.0f, gridSizeX, gridSizeY);
    // diffusion
    float alpha = -1;
    float rBeta = 1 / 4;
    for (int s = 0; s < 20; s++) {
        cuda_jacobi << <numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, alpha, rBeta, u, tmp);
        swap(tmp, u);
        setBoundary << <numberofblocks, numberofthreads >> > (u, 1.0f, gridSizeX, gridSizeY);
    }
    // subGradient
    cuda_subgradient << <numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, u, tmp);
    swap(tmp, u);
    setBoundary << <numberofblocks, numberofthreads >> > (u, -1.0f, gridSizeX, gridSizeY);
    //GPU finish
    hipDeviceSynchronize();
    // apply color
    for (int i = 0; i < gridSizeY; i++) {
        for (int j = 0; j < gridSizeX; j++) {
            pixels[(i * gridSizeX + j) * 4] = 138;
            pixels[(i * gridSizeX + j) * 4 + 1] = 43;
            pixels[(i * gridSizeX + j) * 4 + 2] = 226;
            float amp = sqrtf(u[i * gridSizeX + j].x * u[i * gridSizeX + j].x + u[i * gridSizeX + j].y * u[i * gridSizeX + j].y) * 150;
            if (amp > 255) pixels[(i * gridSizeX + j) * 4 + 3] = 255;
            else pixels[(i * gridSizeX + j) * 4 + 3] = (int) amp;
        }
    }
}

void Solver::print(float4* matrix) {
    for (int i = 0; i < gridSizeY; i++) {
        for (int j = 0; j < gridSizeX; j++) {
            float amp = sqrtf(matrix[i * gridSizeX + j].x * matrix[i * gridSizeX + j].x + matrix[i * gridSizeX + j].y * matrix[i * gridSizeX + j].y);
            std::cout << std::fixed << std::setprecision(0) << amp;
        }
        std::cout << "\n";
    }
}
//main.cpp
int main()
{
    int W = 400, H = 400;
    RenderWindow window(VideoMode(W, H), "test");
    //window.setFramerateLimit(60);

    Uint8* pixels = new Uint8[W * H * 4];
    Texture texture;
    texture.create(W, H);
    Sprite sprite(texture);
    for (register int i = 0; i < W * H * 4; i += 4) {
        pixels[i] = 0;
        pixels[i + 1] = 0;
        pixels[i + 2] = 0;
        pixels[i + 3] = 255;
    }

    Clock clock;
    Time t;
    Vector2i new_pos, old_pos;
    float2 forceVector = make_float2(0, 0);
    float2 forceOrigin = make_float2(0, 0);
    bool click_flag = false;
    Solver stableSolver(W, H, W);
    stableSolver.reset();
    float  timestep = 0.1;
    while (window.isOpen())
    {
        Event event;
        while (window.pollEvent(event))
        {
            switch (event.type) {
            case Event::Closed:
                window.close();
                break;
            case Event::Event::MouseButtonReleased:
                click_flag = false;
                break;
            case Event::MouseButtonPressed:
                click_flag = true;
                old_pos.x = event.mouseButton.x;
                old_pos.y = event.mouseButton.y;
                new_pos.x = event.mouseButton.x;
                new_pos.y = event.mouseButton.y;
                break;
            case Event::MouseMoved:
                if (click_flag) {
                    new_pos.x = event.mouseMove.x;
                    new_pos.y = event.mouseMove.y;
                }
                break;
            default:
                break;
            }

        }


        float elapsed = clock.getElapsedTime().asSeconds();
        if (elapsed > timestep) {
            if (click_flag) {
                forceOrigin = make_float2(old_pos.y, old_pos.x);
                forceVector = make_float2(new_pos.y - old_pos.y, new_pos.x - old_pos.x);
                old_pos = new_pos;
            }
            stableSolver.update(timestep, forceOrigin, forceVector, pixels);
            forceOrigin = make_float2(0, 0);
            forceVector = make_float2(0, 0);
            clock.restart();
        }


        texture.update(pixels);
        window.clear();
        window.draw(sprite);
        window.display();
    }

    return 0;
}
