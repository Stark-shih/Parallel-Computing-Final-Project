#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <SFML/Graphics.hpp>
#include <iostream>
#include <cmath>
#include <assert.h>
#include <iomanip>
#include <stdio.h>

using namespace sf;

//solver.h
class Solver
{
private:
    /* cuda */
    int numberofblocks = 20;
    int numberofthreads = 20;
    /* data */
    int screenWidth;
    int screenHeight;
    int gridSizeX;
    int gridSizeY;
    int deviceId;

    float minX;
    float minY;
    float maxX;
    float maxY;
    float viscosity;

    float4* u;
    float4* tmp;
    float4* div;
    float4* p;
    float4* dye;
    float4* dye_out;
    
public:
    Solver(int width, int height, int resolution);
    ~Solver();
    void reset(const Uint8* pixels);
    void update(float dt, float2 forceOrigin, float2 forceVector);
    void print(float4* matrix);
    unsigned char* pixels;
};
//solver.cpp
Solver::Solver(int screenWidth, int screenHeight, int resolution)
{
    assert((resolution * screenHeight) % screenWidth == 0);
    this->screenWidth = screenWidth;
    this->screenHeight = screenHeight;
    gridSizeX = resolution;
    gridSizeY = resolution * screenHeight / screenWidth;
    minX = 1.0f;
    minY = 1.0f;
    maxX = gridSizeX - 1.0f;
    maxY = gridSizeY - 1.0f;
    viscosity = 1e-6f;
}

Solver::~Solver()
{
}

void Solver::reset(const Uint8* pixels) {
    int numberOfSMs;
    hipGetDevice(&(this->deviceId));
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    this->numberofblocks = 16 * numberOfSMs;
    this->numberofthreads = 128;

    hipMallocManaged(&(this->u), gridSizeY * gridSizeX * sizeof(float4));
    hipMallocManaged(&(this->tmp), gridSizeY * gridSizeX * sizeof(float4));
    hipMallocManaged(&(this->div), gridSizeY * gridSizeX * sizeof(float4));
    hipMallocManaged(&(this->p), gridSizeY * gridSizeX * sizeof(float4));
    hipMallocManaged(&(this->dye), gridSizeY * gridSizeX * sizeof(float4));
    hipMallocManaged(&(this->dye_out), gridSizeY * gridSizeX * sizeof(float4));
    hipMallocManaged(&(this->pixels), 4*gridSizeY * gridSizeX * sizeof(unsigned char));

    for (int i = 0; i < gridSizeY; i++) {
        for (int j = 0; j < gridSizeX; j++) {
            dye[i * gridSizeX + j].x = pixels[(i * gridSizeX + j) * 4];
            dye[i * gridSizeX + j].y = pixels[(i * gridSizeX + j) * 4 + 1];
            dye[i * gridSizeX + j].z = pixels[(i * gridSizeX + j) * 4 + 2];
            dye[i * gridSizeX + j].w = pixels[(i * gridSizeX + j) * 4 + 3];
        }
    }
    hipMemset(this->u, 0, gridSizeY * gridSizeX * sizeof(float4));
    hipMemset(this->tmp, 0, gridSizeY * gridSizeX * sizeof(float4));
    hipMemset(this->div, 0, gridSizeY * gridSizeX * sizeof(float4));
    hipMemset(this->p, 0, gridSizeY * gridSizeX * sizeof(float4));
    hipMemset(this->dye_out, 0, gridSizeY * gridSizeX * sizeof(float4));
}
void swap(float4*& field1, float4*& field2) {
    float4* temp = field1;
    field1 = field2;
    field2 = temp;
}
// __device__
float _clampTo_0_1(float val) {
    if (val < 0.f) val = 0;
    if (val > 255.0f) val = 255;
    return val;
}
__global__
void setBoundary(float4* field, float sc, int w, int h) {
    /* horizontal: the first line and the last line */
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int j = index; j < w; j += stride) {
        field[j] = make_float4(sc * field[w + j].x, sc * field[w + j].y, sc * field[w + j].z, sc * field[w + j].w);
        field[(h - 1) * w + j] = make_float4(sc * field[(h - 2) * w + j].x, sc * field[(h - 2) * w + j].y, sc * field[(h - 2) * w + j].z, sc * field[(h - 2) * w + j].w);
    }
    /* vetrtical */
    for (int i = index; i < h; i += stride) {
        field[i * w] = make_float4(sc * field[i * w + 1].x, sc * field[i * w + 1].y, sc * field[i * w + 1].z, sc * field[i * w + 1].w);
        field[i * w + w - 1] = make_float4(sc * field[i * w + w - 2].x, sc * field[i * w + w - 2].y, sc * field[i * w + w - 2].z, sc * field[i * w + w - 2].w);
    }
}
__global__
void cuda_addForce(int gridSizeX, int gridSizeY, float2 forceOrigin, float2 forceVector, float4* w_in, float4* w_out) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;
        float2 pos = make_float2(b, a);

        float distance = sqrtf((pos.x - forceOrigin.x) * (pos.x - forceOrigin.x) + (pos.y - forceOrigin.y) * (pos.y - forceOrigin.y));
        float amp = exp(-distance / 10);
        // amp = (amp);
        w_out[a * gridSizeX + b].x = (w_in[a * gridSizeX + b].x + forceVector.x * amp);
        w_out[a * gridSizeX + b].y = (w_in[a * gridSizeX + b].y + forceVector.y * amp);
    }
}
__global__
void cuda_advect(int gridSizeX, int gridSizeY, float dt, float4* u, float4* xNew) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    float oldx, oldy, dx, dy, mdx, mdy;
    int xid0, xid1, yid0, yid1;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;
        oldx = b - dt * u[a * gridSizeX + b].x * gridSizeX;
        oldy = a - dt * u[a * gridSizeY + b].y * gridSizeX;
        oldx = fmax(0.5f, fmin(gridSizeX - 0.5f, oldx));
        oldy = fmax(0.5f, fmin(gridSizeY - 0.5f, oldy));
        xid0 = (int)oldx;
        xid1 = xid0 + 1;
        yid0 = (int)oldy;
        yid1 = yid0 + 1;
        dx = oldx - xid0;
        mdx = 1 - dx;
        dy = oldy - yid0;
        mdy = 1 - dy;
        xNew[a * gridSizeX + b].x = mdx * (mdy * u[yid0 * gridSizeX + xid0].x + dy * u[yid1 * gridSizeX + xid0].x) + dx * (mdy * u[yid0 * gridSizeX + xid1].x + dy * u[yid1 * gridSizeX + xid1].x);
        xNew[a * gridSizeX + b].y = mdx * (mdy * u[yid0 * gridSizeX + xid0].y + dy * u[yid1 * gridSizeX + xid0].y) + dx * (mdy * u[yid0 * gridSizeX + xid1].y + dy * u[yid1 * gridSizeX + xid1].y);
        // xNew[a * gridSizeX + b].x = (xNew[a * gridSizeX + b].x / 1.005);
        // xNew[a * gridSizeX + b].y = (xNew[a * gridSizeX + b].y / 1.005);

    }
}
__global__
void cuda_divergence(int gridSizeX, int gridSizeY, float4* w, float4* div, float4* p) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;

        float wL = w[a * gridSizeX + b - 1].x;
        float wR = w[a * gridSizeX + b + 1].x;
        float wT = w[(a - 1) * gridSizeX + b].y;
        float wB = w[(a + 1) * gridSizeX + b].y;
        div[a * gridSizeX + b].w = -0.5 * ((wR - wL) + (wB - wT));// / gridSizeX;
        p[a * gridSizeX + b] = make_float4(0, 0, 0, 0);
    }
}
__global__
void cuda_jacobi(int gridSizeX, int gridSizeY, float alpha, float beta, float4* x, float4* b_, float4* xNew) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;

        float4 xL = x[a * gridSizeX + b - 1];
        float4 xR = x[a * gridSizeX + b + 1];
        float4 xT = x[(a - 1) * gridSizeX + b];
        float4 xB = x[(a + 1) * gridSizeX + b];
        float4 bc = b_[a * gridSizeX + b];
        xNew[a * gridSizeX + b].z = ((xL.z + xR.z + xT.z + xB.z) * alpha + bc.w) / beta;
    }
}
__global__
void cuda_subgradient(int gridSizeX, int gridSizeY, float4* p, float4* w, float4* uNew) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;

        float4 pL = p[a * gridSizeX + b - 1];
        float4 pR = p[a * gridSizeX + b + 1];
        float4 pT = p[(a - 1) * gridSizeX + b];
        float4 pB = p[(a + 1) * gridSizeX + b];

        uNew[a * gridSizeX + b] = w[a * gridSizeX + b];
        uNew[a * gridSizeX + b].x -= 0.5 * (pR.z - pL.z);// * gridSizeX;
        uNew[a * gridSizeX + b].y -= 0.5 * (pB.z - pT.z);// * gridSizeX;
    }
}
__global__
void cuda_DyeAdvect(int gridSizeX, int gridSizeY, float dt, float4* u, float4* dye, float4* dye_out) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    float oldx, oldy, dx, dy, mdx, mdy;
    int xid0, xid1, yid0, yid1;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        //if (a == 0 || a == gridSizeY - 1 || b == 0 || b == gridSizeX - 1) continue;
        oldx = b - dt * u[a * gridSizeX + b].x * gridSizeX;
        oldy = a - dt * u[a * gridSizeY + b].y * gridSizeX;
        oldx = fmax(0.5f, fmin(gridSizeX - 0.5f, oldx));
        oldy = fmax(0.5f, fmin(gridSizeY - 0.5f, oldy));
        xid0 = (int)oldx;
        xid1 = xid0 + 1;
        yid0 = (int)oldy;
        yid1 = yid0 + 1;
        dx = oldx - xid0;
        mdx = 1 - dx;
        dy = oldy - yid0;
        mdy = 1 - dy;
        dye_out[a * gridSizeX + b].x = mdx * (mdy * dye[yid0 * gridSizeX + xid0].x + dy * dye[yid1 * gridSizeX + xid0].x) + dx * (mdy * dye[yid0 * gridSizeX + xid1].x + dy * dye[yid1 * gridSizeX + xid1].x);
        dye_out[a * gridSizeX + b].y = mdx * (mdy * dye[yid0 * gridSizeX + xid0].y + dy * dye[yid1 * gridSizeX + xid0].y) + dx * (mdy * dye[yid0 * gridSizeX + xid1].y + dy * dye[yid1 * gridSizeX + xid1].y);
        dye_out[a * gridSizeX + b].z = mdx * (mdy * dye[yid0 * gridSizeX + xid0].z + dy * dye[yid1 * gridSizeX + xid0].z) + dx * (mdy * dye[yid0 * gridSizeX + xid1].z + dy * dye[yid1 * gridSizeX + xid1].z);
    }
}
__global__
void dye_pixels(int gridSizeX, int gridSizeY, float4* dye, unsigned char* pixels) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < gridSizeY * gridSizeX; i += stride) {
        int a = i / gridSizeX;
        int b = i - a * gridSizeX;
        pixels[(a * gridSizeX + b) * 4] = dye[a * gridSizeX + b].x;
        pixels[(a * gridSizeX + b) * 4 + 1] = dye[a * gridSizeX + b].y;
        pixels[(a * gridSizeX + b) * 4 + 2] = dye[a * gridSizeX + b].z;
        pixels[(a * gridSizeX + b) * 4 + 3] = 255;
    }
}
__global__
void cuda_print(float4* u) {
    printf("%f\n", u[200 * 400 + 200].z);
}
//adect->forceaply->applyDye->divergence->jacobiviscousdiffusion->applygradient
void Solver::update(float dt, float2 forceOrigin, float2 forceVector) {


    // external force
    cuda_addForce << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, forceOrigin, forceVector, u, tmp);
    swap(tmp, u);
    setBoundary << < numberofblocks, numberofthreads >> > (u, -1.0f, gridSizeX, gridSizeY);
    // diffussion
    for (int s = 0; s < 15; s++) {
        cuda_jacobi << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, dt * viscosity * gridSizeX * gridSizeY, 1 + 4 * dt * viscosity * gridSizeX * gridSizeY, u, u, tmp);
        cuda_jacobi << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, dt * viscosity * gridSizeX * gridSizeY, 1 + 4 * dt * viscosity * gridSizeX * gridSizeY, tmp, tmp, u);
        // swap(tmp, u);
        setBoundary << < numberofblocks, numberofthreads >> > (u, -1.0f, gridSizeX, gridSizeY);
    }
    // -------------------- projection start----------------------
    // divergence
    cuda_divergence << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, u, div, p);
    setBoundary << <numberofblocks, numberofthreads >> > (div, 1.0f, gridSizeX, gridSizeY);
    // pressure
    for (int s = 0; s < 20; s++) {
        cuda_jacobi << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, 1, 4, p, div, tmp);
        cuda_jacobi << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, 1, 4, tmp, div, p);
        setBoundary << < numberofblocks, numberofthreads >> > (p, 1.0f, gridSizeX, gridSizeY);
    }
    // subGradient
    cuda_subgradient << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, p, u, tmp);
    swap(tmp, u);
    setBoundary << < numberofblocks, numberofthreads >> > (u, -1.0f, gridSizeX, gridSizeY);
    // -------------------- projection end ----------------------
    // advect
    cuda_advect << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, dt, u, tmp);
    swap(tmp, u);
    setBoundary << < numberofblocks, numberofthreads >> > (u, -1.0f, gridSizeX, gridSizeY);
    // -------------------- projection start----------------------
    // divergence
    cuda_divergence << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, u, div, p);
    setBoundary << <numberofblocks, numberofthreads >> > (div, 1.0f, gridSizeX, gridSizeY);
    // pressure
    for (int s = 0; s < 20; s++) {
        cuda_jacobi << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, 1, 4, p, div, tmp);
        cuda_jacobi << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, 1, 4, tmp, div, p);
        setBoundary << < numberofblocks, numberofthreads >> > (p, 1.0f, gridSizeX, gridSizeY);
    }
    // subGradient
    cuda_subgradient << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, p, u, tmp);
    swap(tmp, u);
    setBoundary << < numberofblocks, numberofthreads >> > (u, -1.0f, gridSizeX, gridSizeY);
    // -------------------- projection end ----------------------
    //dye
    cuda_DyeAdvect << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, dt, u, dye, dye_out);
    swap(dye_out, dye);
    // apply color
    dye_pixels << < numberofblocks, numberofthreads >> > (gridSizeX, gridSizeY, dye, pixels);
    hipDeviceSynchronize();
    //finish
    hipMemPrefetchAsync(pixels, 4* gridSizeY * gridSizeX * sizeof(unsigned char), deviceId);

}

void Solver::print(float4* matrix) {
    for (int i = 0; i < gridSizeY; i++) {
        for (int j = 0; j < gridSizeX; j++) {
            float amp = sqrtf(matrix[i * gridSizeX + j].x * matrix[i * gridSizeX + j].x + matrix[i * gridSizeX + j].y * matrix[i * gridSizeX + j].y);
            std::cout << std::fixed << std::setprecision(0) << amp;
        }
        std::cout << "\n";
    }
}
//main.cpp
int main()
{
    int W = 400, H = 400;
    RenderWindow window(VideoMode(W, H), "go down");
    //window.setFramerateLimit(60);

    Texture texture;
    texture.create(W, H);
    Image img;
    img.loadFromFile("../images.jpg");
    Sprite sprite(texture);
    Clock clock;
    Time t;
    Vector2i last_pos, now_pos;
    float2 forceVector = make_float2(0, 0);
    float2 forceOrigin = make_float2(0, 0);
    bool click_flag = false;
    Solver stableSolver(W, H, W);
    stableSolver.reset(img.getPixelsPtr());
    float  timestep = 0.01;
    while (window.isOpen())
    {
        Event event;
        while (window.pollEvent(event))
        {
            switch (event.type) {
            case Event::Closed:
                window.close();
                break;
            case Event::Event::MouseButtonReleased:
                click_flag = false;
                forceOrigin = make_float2(0, 0);
                forceVector = make_float2(0, 0);
                break;
            case Event::MouseButtonPressed:
                click_flag = true;
                last_pos.x = event.mouseButton.x;
                last_pos.y = event.mouseButton.y;
                break;
            case Event::MouseMoved:
                if (click_flag) {
                    now_pos.x = event.mouseMove.x;
                    now_pos.y = event.mouseMove.y;
                    forceOrigin = make_float2(last_pos.x, last_pos.y);
                    forceVector = make_float2(now_pos.x - last_pos.x, now_pos.y - last_pos.y);
                    last_pos.x = now_pos.x;
                    last_pos.y = now_pos.y;
                }
                break;
            default:
                break;
            }

        }


        float elapsed = clock.getElapsedTime().asSeconds();
        if (elapsed > timestep) {
            stableSolver.update(timestep, forceOrigin, forceVector);
            clock.restart();
        }


        texture.update(stableSolver.pixels);
        window.clear();
        window.draw(sprite);
        window.display();
    }

    return 0;
}
